
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void p1(int* indices, int n, int* result){
    
    for(int i = threadIdx.x; i < n; i+=blockDim.x){
        result[i] = indices[i+1] - indices[i];
    }
}   



__global__ void p2_3(int* p1, int* c3, int* indices, int* data, int n, int* result1, int* result2){
    
    for(int i = threadIdx.x; i < n; i+=blockDim.x){
        result1[i] = 0;
        result2[i] = 0;
        for(int j = indices[i]; j < indices[i+1]; j++){
            result1[i] += p1[data[j]];
        }
        result1[i]-=(p1[i]+2*c3[i]);
        result2[i] = p1[i]*(p1[i]-1)/2-c3[i];
    }
    
}

__global__ void c3(int* indices, int n, int* data, int* result){
    int col;
    for(int i = threadIdx.x; i < n; i+=blockDim.x){//All nodes
        if(i<n){
            result[i] = 0;
            int start_i = indices[i];
            int end_i = indices[i+1];
            int width_i = end_i-start_i;
            for(int j = start_i; j < end_i; j++){//CSC
                col = data[j];
                int start_j = indices[col];
                int width_j = indices[col+1] - start_j;
                int idx_i = 0;
                int idx_j = 0;
                while(idx_i<width_i&&idx_j<width_j)//matrix mult
                {


                    if(data[start_i+idx_i]==data[start_j+idx_j]){
                        result[i]++;
                        idx_i++;
                        idx_j++;
                        continue;
                    }
                    if(data[start_i+idx_i]>data[start_j+idx_j]){
                        idx_j++;

                        continue;
                    }
                    else{

                        idx_i++;
                    }
                }
            }
            result[i] = result[i]/2;
        }
    }
}


int main(int argc, char const *argv[])
{
    int x[7] = {0,2,6,9,13,17,18};
    int y[18] = {1,4,0,2,3,4,1,3,4,1,2,4,5,0,1,2,3,3};
    int* d1;
    int* d2;
    int* d3;
    int* d4;
    int* ind;
    int* data;
    int* host1 = (int*)malloc(sizeof(int)*6);
    int* host2 = (int*)malloc(sizeof(int)*6);

    hipMalloc((void**)&ind,7*sizeof(int));
    hipMalloc((void**)&data,18*sizeof(int));
    hipMemcpy(ind,&x[0],7*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(data,&y[0],18*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&d1,6*sizeof(int));
    hipMalloc((void**)&d2,6*sizeof(int));
    hipMalloc((void**)&d3,6*sizeof(int));
    hipMalloc((void**)&d4,6*sizeof(int));

    
    p1<<<1,1024>>>(ind,6,d1);

    c3<<<1,1024>>>(ind,6,data,d4);

    p2_3<<<1,2>>>(d1,d4,ind,data,6,d2,d3);

    hipMemcpy(host1,d2,6*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(host2,d3,6*sizeof(int),hipMemcpyDeviceToHost);

    for(int i = 0; i < 6; i++){
        printf("%d %d\n",host1[i], host2[i]);
    }
    //int** l = p2_3(host,m,&x[0],&y[0],6);
    

    // for(int i = 0; i < 6; i ++){
    //     printf("%d %d %d %d\n",host[i],l[0][i],l[1][i], m[i]);
    // }
 
    hipFree(d1);
    hipFree(d4);

    return 0;
}
