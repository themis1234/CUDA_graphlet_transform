
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void p1(int* indices, int n, int* result){
    for(int i = threadIdx.x; i < n; i+=blockDim.x){
        result[i] = indices[i+1] - indices[i];
    }
}   



__global__ void p2_3(int* p1, int* c3, int* indices, int* data, int n, int* result1, int* result2){
    
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i+=blockDim.x * gridDim.x){
        result1[i] = 0;
        result2[i] = 0;
        for(int j = indices[i]; j < indices[i+1]; j++){
            result1[i] += p1[data[j]];
        }
        result1[i]-=(p1[i]+2*c3[i]);
        result2[i] = p1[i]*(p1[i]-1)/2-c3[i];
    }
    
}

__global__ void c3(int* indices, int n, int* data, int* result){
    int col;
    for(int i = threadIdx.x; i < n; i+=blockDim.x){//All nodes
        if(i<n){
            result[i] = 0;
            int start_i = indices[i];
            int end_i = indices[i+1];
            int width_i = end_i-start_i;
            for(int j = start_i; j < end_i; j++){//CSC
                col = data[j];
                int start_j = indices[col];
                int width_j = indices[col+1] - start_j;
                int idx_i = 0;
                int idx_j = 0;
                while(idx_i<width_i&&idx_j<width_j)//matrix mult
                {


                    if(data[start_i+idx_i]==data[start_j+idx_j]){
                        result[i]++;
                        idx_i++;
                        idx_j++;
                        continue;
                    }
                    if(data[start_i+idx_i]>data[start_j+idx_j]){
                        idx_j++;

                        continue;
                    }
                    else{

                        idx_i++;
                    }
                }
            }
            result[i] = result[i]/2;
        }
    }
}


int main(int argc, char const *argv[])
{
    

    FILE* ind;
    int num_ind;
    ind = fopen("indices.txt","r");
    fscanf(ind,"%d\n",&num_ind);
    int* host_ind = (int*)malloc(sizeof(int)*num_ind);
    int* indices;
    hipMalloc((void**)&indices,num_ind*sizeof(int));

    for(int i = 0; i < num_ind; i++){
        fscanf(ind,"%d\n",&host_ind[i]);
    }    
    hipMemcpy(indices,host_ind,num_ind*sizeof(int),hipMemcpyHostToDevice);


    int num_data;
    ind = fopen("data.txt","r");
    fscanf(ind,"%d\n",&num_data);
    int* host_data = (int*)malloc(sizeof(int)*num_data);
    int* data; 
    hipMalloc((void**)&data,num_data*sizeof(int));

    for(int i = 0; i < num_data; i++){
        fscanf(ind,"%d\n",&host_data[i]);
    }    
    hipMemcpy(data,host_data,num_data*sizeof(int),hipMemcpyHostToDevice);
    int n = num_ind-1;

    int* d1;
    int* d2;
    int* d3;
    int* d4;

    hipMalloc((void**)&d1,n*sizeof(int));
    hipMalloc((void**)&d2,n*sizeof(int));
    hipMalloc((void**)&d3,n*sizeof(int));
    hipMalloc((void**)&d4,n*sizeof(int));

    printf("Data copied for host to device, Finding Nodes' degree...\n");
    p1<<<1,1024>>>(indices,n,d1);
    hipDeviceSynchronize();

    printf("Nodes' degree found, Calculating 3-cliques...\n");

    c3<<<4,1024>>>(indices,n,data,d4);
    hipDeviceSynchronize();
    printf("3-cliques found, Calculating 2-paths and biforks...\n");



    p2_3<<<1,1024>>>(d1,d4,indices,data,n,d2,d3);
    hipDeviceSynchronize();
    printf("All Done\n");


    // cudaMemcpy(host1,d1,n*sizeof(int),cudaMemcpyDeviceToHost);
    // cudaMemcpy(host2,d4,n*sizeof(int),cudaMemcpyDeviceToHost);

    // for(int i = 0; i < n; i++){
    //     printf("%d %d %d\n",host1[i], host2[i],i);
    // }
    //int** l = p2_3(host,m,&x[0],&y[0],6);
    

    // for(int i = 0; i < 6; i ++){
    //     printf("%d %d %d %d\n",host[i],l[0][i],l[1][i], m[i]);
    // }
 
    hipFree(d1);
    hipFree(d2);
    hipFree(d3);
    hipFree(d4);

    return 0;
}
